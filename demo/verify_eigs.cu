#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "../HLanc.hpp"

struct eigs_t {
	int nev;
	int n;
	double * value;
	double * vector;
	eigs_t ( char const * filename ) {
		FILE * fp = fopen ( filename, "r" );
		if ( !fp ) {
			HLanc::warn ( "Cannot open file \"%s\": %m\n", filename );
			exit ( 1 );
		}

		if ( 1 != fscanf ( fp, "%d", &nev ) ) {
			HLanc::warn ( "Error reading nev from \"%s\".\n", filename );
			exit ( 1 );
		}
		if ( 1 != fscanf ( fp, "%d", &n ) ) {
			HLanc::warn ( "Error reading n from \"%s\".\n", filename );
			exit ( 1 );
		}

		value  = new double[nev];
		vector = new double[n * nev];
		for ( int i = 0; i < nev; ++i ) {
			if ( 1 != fscanf ( fp, "%lE", &value[i] ) ) {
				HLanc::warn ( "Error reading eig value at index %d from \"%s\".\n", i, filename );
				exit ( 1 );
			}
		}
		for ( int r = 0; r < n; ++r ) {
			for ( int c = 0; c < nev; ++c ) {
				if ( 1 != fscanf ( fp, "%lE", &vector[c * n + r] ) ) {
					HLanc::warn ( "Error reading eig vector at index <%d,%d> from \"%s\".\n", r, c, filename );
					exit ( 1 );
				}
			}
		}

		fclose ( fp );
	}
	~eigs_t ( ) {
		delete [] value;
		delete [] vector;
	}
};

double rdiff ( double const * x, double const * y, int len ) {
	double maxdiff = 0;
	double maxabs = 0;
	for ( int i = 0; i < len; ++i ) {
		double d1 = x[i];
		double d2 = y[i];
		double diff = d1 - d2;
		if ( diff < 0 ) diff = -diff;
		maxabs = max ( maxabs, abs ( d1 ) );
		maxabs = max ( maxabs, abs ( d2 ) );
		if ( maxdiff < diff ) maxdiff = diff;
	}
	if ( maxabs ) maxdiff /= maxabs;
	return maxdiff;
}

int main ( int argc, char * argv[] ) {
	if ( 4 != argc && 5 != argc ) {
		HLanc::warn ( "Usage: %s <bin|mm|pp> <matrix_file> <eigs_file> [dev=0]\n", argv[0] );
		return 1;
	}

	char const *   format = argv[1];
	char const * mat_file = argv[2];
	HLanc::dcsr_matrix_t m ( format, mat_file );

	eigs_t eigs ( argv[3] );
	int dev = 0;
	if ( 4 < argc ) { dev = atoi ( argv[4] ); }

	int nev = eigs.nev;
	int n   = eigs.n;

	HLanc::dcsrmv_operator_gpu op ( m, dev );

	double * x = new double[n];
	double * y = new double[n];

	for ( int c = 0; c < nev; ++c ) {
		double   value  = eigs.value[c];
		double * vector = eigs.vector + c * n;
		op ( vector, x );
		for ( int r = 0; r < n; ++r ) {
			y[r] = vector[r] * value;
		}
		HLanc::warn ( "max r-diff of vector[%3d]: %.6lE\n", c, rdiff ( x, y, n ) );
	}

	delete [] y;
	delete [] x;
	return 0;
}
